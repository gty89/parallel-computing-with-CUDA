/*
Collatz code for CS 4380 / CS 5351

Copyright (c) 2020 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdio>
#include <algorithm>
#include <sys/time.h>
#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 512;

static __global__ void collatz(const long bound, int* const maxlen)
{
  // compute sequence lengths
  const long i = threadIdx.x + blockIdx.x * (long)blockDim.x + 1;
  if( i < bound) {
    long val = i;
    int len = 1;
    while (val != 1) {
      len++;
      if ((val % 2) == 0) {
        val = val / 2;  // even
      } else {
        val = 3 * val + 1;  // odd
      }
    }
  // atomicMax( maxlen, max( maxlen, len));
  if( len > *maxlen)
    atomicMax( maxlen, len); 
  }
  return;
}


static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Collatz v1.3\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s upper_bound\n", argv[0]); exit(-1);}
  const long bound = atol(argv[1]);
  if (bound < 2) {fprintf(stderr, "ERROR: upper_bound must be at least 2\n"); exit(-1);}
  printf("upper bound: %ld\n", bound);
  
  int maxlen = 0;
  int *d_maxlen;
  int size = sizeof(int);
  if (hipSuccess != hipMalloc((void**) &d_maxlen, size)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  if (hipSuccess != hipMemcpy(d_maxlen, &maxlen, size, hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR: copying to device failed\n"); exit(-1);}

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
  collatz<<<(bound + ThreadsPerBlock - 2)/ThreadsPerBlock, ThreadsPerBlock>>>(bound, d_maxlen); 
  // I use -2 instead of -1 because of more precise. For instance, if bound==11 and Threads==5, we only use 10 threads to conputer bound from 1 to 10, so we need only 2 blocks instead of 3
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  CheckCuda();
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.5f s\n", runtime);

  // print result
  if (hipSuccess != hipMemcpy(&maxlen, d_maxlen, size, hipMemcpyDeviceToHost)) {fprintf(stderr, "ERROR: copying from device failed\n"); exit(-1);}
  printf("longest sequence: %d elements\n", maxlen);
  hipFree( d_maxlen);

  return 0;
}
