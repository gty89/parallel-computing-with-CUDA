#include "hip/hip_runtime.h"
/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2020 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "BMP43805351.h"

static const int ThreadsPerBlock = 512;

static __global__ void fractal(const int width, const int frames, unsigned char* const pic)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  if( i < frames * width * width){
    const float Delta = 0.01f;
    const float xMid = 0.28017435f;
    const float yMid = 0.60538436f;
  
    const int col = i % width;
    const int row = (i / width) % width;
    const int frame = i / (width * width);
    // compute pixels of each frame
    const float delta = Delta * powf( 0.981f, frame);
    const float xMin = xMid - delta;
    const float yMin = yMid - delta;
    const float dw = 2.0f * delta / width;
    const float cy = yMin + row * dw;
    const float cx = xMin + col * dw;
    float x = cx;
    float y = cy;
    float x2, y2;
    int depth = 256;
    do {
      x2 = x * x;
      y2 = y * y;
      y = 2.0f * x * y + cy;
      x = x2 - y2 + cx;
      depth--;
    } while ((depth > 0) && ((x2 + y2) < 5.0f));
        pic[frame * width * width + row * width + col] = (unsigned char)depth;
  }
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Fractal v2.0\n");

  // check command line
  if (argc != 3) {fprintf(stderr, "USAGE: %s frame_width number_of_frames\n", argv[0]); exit(-1);}
  const int width = atoi(argv[1]);
  if (width < 10) {fprintf(stderr, "ERROR: frame_width must be at least 10\n"); exit(-1);}
  const int frames = atoi(argv[2]);
  if (frames < 1) {fprintf(stderr, "ERROR: number_of_frames must be at least 1\n"); exit(-1);}
  printf("width: %d\n", width);
  printf("frames: %d\n", frames);

  // allocate picture array
  unsigned char* pic = new unsigned char [frames * width * width];
  unsigned char* d_pic;
  if (hipSuccess != hipMalloc((void **)&d_pic, sizeof(unsigned char) * frames * width * width)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
  fractal<<< (frames * width * width + ThreadsPerBlock - 1)/ThreadsPerBlock, ThreadsPerBlock>>>(width, frames, d_pic);
  hipDeviceSynchronize();
  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0f;
  printf("compute time: %.5f s\n", runtime);
  CheckCuda();
  if (hipSuccess != hipMemcpy( pic, d_pic, sizeof(unsigned char) * frames * width * width, hipMemcpyDeviceToHost)) {fprintf(stderr, "ERROR: copying from device failed\n"); exit(-1);}
  // write result to BMP files
  if ((width <= 256) && (frames <= 60)) {
    for (int frame = 0; frame < frames; frame++) {
      BMP24 bmp(0, 0, width, width);
      for (int y = 0; y < width; y++) {
        for (int x = 0; x < width; x++) {
          const int p = pic[frame * width * width + y * width + x];
          const int blue = 128 + sin((p + 85) * 0.0245436926f) * 127;
          const int green = 128 + sin((p + 0) * 0.0245436926f) * 127;
          const int red = 128 + sin((p + 171) * 0.0245436926f) * 127;
          bmp.dot(x, y, red * 0x010000 + green * 0x000100 + blue * 0x000001);
        }
      }
      char name[32];
      sprintf(name, "fractal%d.bmp", frame + 10000);
      bmp.save(name);
    }
  }

  // clean up
  delete [] pic;
  hipFree(d_pic);
  return 0;
}
